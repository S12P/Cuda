
#include <hip/hip_runtime.h>
#include<stdlib.h>
#include<stdio.h>
const int N = 32;

__global__ void mul(int* A, int* B, int* C){
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int lig = blockIdx.y * blockDim.y + threadIdx.y;

    int index = lig * N + col;
    
    if (col < N && lig < N){
        int inter = 0;
        for (int i = 0; i<N; ++i){
            inter += A[lig*N + i] * B[i*N + col];
        }
        C[index] = inter;
    }
}

__host__ void affiche(int *A, int z){
    for( int i=0;i<z;i++){
        for (int j=0; j<z;j++){
            printf(" %d ",A[i*z+j]);
        }
        printf("\n");
    }
}

int main(void){
    int *A, *B, *C, *da, *db, *dc;
    int size = N * N * sizeof(int);


    hipMalloc((void **) & da, size);
    hipMalloc((void **) & db, size);
    hipMalloc((void **) & dc, size);

    A = (int *)malloc(size);
    B = (int *)malloc(size);
    C = (int *)malloc(size);

    for (int i=0; i<N * N; ++i){
        A[i]=1; B[i]=1;
    }



    hipMemcpy(da, A, size, hipMemcpyHostToDevice);
    hipMemcpy(db, B, size, hipMemcpyHostToDevice);

    //dim3 dimBlock(N, N);
    dim3 dimGrid(N, N);

    mul<<<dimGrid, dimGrid>>>(da, db, dc);

    hipMemcpy(C, dc, size, hipMemcpyDeviceToHost);

    affiche(C, N);


    free(A); free(B); free(C);
    hipFree(da); hipFree(db); hipFree(dc);
    return 0;
}
