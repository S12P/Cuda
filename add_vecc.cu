#include<stdio.h>
#include<stdlib.h>
#include <iostream>
#include<hip/hip_runtime.h>
#define N (1024*1024)
#define nb_thread 512
__global__ void add(int *a, int *b, int *c) {
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	c[index] = a[index] + b[index];
}


int main(void) {
	int *a,*b,*c; // sur le cpu
	int *d_a, *d_b, *d_c; // sur le gpu
	
	int size = N * sizeof(int);
	
	hipMalloc((void **) & d_a, size);
	hipMalloc((void **) & d_b, size);
	hipMalloc((void **) & d_c, size);


	a = (int *)malloc(size);
	b = (int *)malloc(size);
	c = (int *)malloc(size);
	for (int i = 0; i<N; i++){
		a[i]=rand()%20;
		b[i]=rand()%20;
	}
	

	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

	add<<<N/nb_thread,nb_thread>>>(d_a, d_b, d_c);

	hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
	free(a);
	free(b);
	free(c);
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	for (int i=0; i<N;i++)	
		printf("%d %d %d \n",a[i],b[i],c[i]);
	return 0;
}

