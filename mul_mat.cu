
#include <hip/hip_runtime.h>
#include<stdlib.h>
#include<stdio.h>
const int N = 32;
const int blocksize = 16;

__global__ void mul(int* A, int* B, int* C){
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int lig = blockIdx.y * blockDim.y + threadIdx.y;

    int index = lig * N + col;
    
    if (col < N && lig < N){
        int inter = 0;
        for (int i = 0; i<N; ++i){
            inter += A[lig*N + i] * B[i*N + col];
        }
        C[index] = inter;
    }
}

__host__ void affiche(int *A, int z){
    for( int i=0;i<z;i++){
        for (int j=0; j<z;j++){
            printf(" %d ",A[i*z+j]);
        }
        printf("\n");
    }
}

int main(void){
    int *A, *B, *C, *da, *db, *dc;
    int size = N * N * sizeof(int);


    hipMalloc((void **) & da, size);
    hipMalloc((void **) & db, size);
    hipMalloc((void **) & dc, size);

    A = (int *)malloc(size);
    B = (int *)malloc(size);
    C = (int *)malloc(size);

    for (int i=0; i<N * N; ++i){
        A[i]=1; B[i]=1;
    }



    hipMemcpy(da, A, size, hipMemcpyHostToDevice);
    hipMemcpy(db, B, size, hipMemcpyHostToDevice);

    dim3 dimBlock(blocksize, blocksize);
    dim3 dimGrid(N*N/dimBlock.x, N*N/dimBlock.y);

    mul<<<dimGrid, dimBlock>>>(da, db, dc);

    hipMemcpy(C, dc, size, hipMemcpyDeviceToHost);

    affiche(C, N);


    free(A); free(B); free(C);
    hipFree(da); hipFree(db); hipFree(dc);
    return 0;
}

